#include <iostream>
#include <fstream>
#include "Eigen.h"
#include "VirtualSensor.h"
#include "SimpleMesh.h"
#include "SurfaceMeasurement.h"
#include "SurfacePredictionCuda.h"
#include "CudaICPOptimizer.h"
#include "BilateralFilter.h"
#include "VolumetricGridCuda.h"

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>

#include "VolumetricGridCuda.h"


#define USE_GPU_ICP	1
#define USE_REDUCTION_ICP 1
#define SHOW_MESH 0


int reconstructRoom() {
    // Setup virtual sensor
    std::string filenameIn = std::string("../../data/rgbd_dataset_freiburg1_xyz/");
    std::string filenameBaseOut = std::string("../../outputs/mesh_");


    // Load video
	std::cout << "Initialize virtual sensor..." << std::endl;
	VirtualSensor sensor;
	if (!sensor.init(filenameIn)) {
		std::cout << "Failed to initialize the sensor!\nCheck file path!" << std::endl;
		return -1;
	}

    const unsigned depthFrameWidth = sensor.getDepthImageWidth();
    const unsigned depthFrameHeight = sensor.getDepthImageHeight();
    const size_t N = depthFrameWidth * depthFrameHeight;

    // Setup the ICP optimizer.
    ICPOptimizer* optimizer;
    if(USE_REDUCTION_ICP) {
        optimizer = new LinearICPCubOptimizer(depthFrameWidth,depthFrameHeight);
    } else {
        optimizer = new LinearICPCudaOptimizer(depthFrameWidth,depthFrameHeight);
    }

    optimizer->setMatchingMaxDistance(0.1f);
    //optimizer->setMatchingMaxDistance(0.0003f);
    optimizer->setMatchingMaxAngle(1.0472f); // 0.523599 // 1.0472f
    optimizer->usePointToPlaneConstraints(true);
    optimizer->setNbOfIterations(20);

    // Intrinsics on host memory
    Matrix3f depthIntrinsics = sensor.getDepthIntrinsics();

    // Intrinsics on device memory
    Matrix3f *cudaDepthIntrinsics;
    CUDA_CALL(hipMalloc((void **) &cudaDepthIntrinsics, sizeof(Matrix3f)));
    CUDA_CALL(hipMemcpy(cudaDepthIntrinsics, depthIntrinsics.data(), sizeof(Matrix3f), hipMemcpyHostToDevice));


    Matrix4f currentCameraToWorld = Matrix4f::Identity();
    Matrix4f base_pose_cpu = Matrix4f::Identity();

	// We store the estimated camera poses. [on Host memory]
	// estimated poses will save world to camera pose
	std::vector<Matrix4f> estimatedPoses;

    TransformHelper transformHelper;

    SurfaceMeasurement surfaceMeasurement(depthIntrinsics.inverse(), 0);
    VolumetricGridCuda volumetricGrid(cudaDepthIntrinsics,  &base_pose_cpu);
    SurfacePredictionCuda surfacePrediction(cudaDepthIntrinsics, 0);

    // Defining memory for previous and current frames,  [on Device memory]
    FrameData previousFrame;
    FrameData currentFrame;

    float *unfilteredDepth;

    previousFrame.width =  depthFrameWidth;
    previousFrame.height = depthFrameHeight;

    currentFrame.width =  depthFrameWidth;
    currentFrame.height = depthFrameHeight;

    CUDA_CALL(hipMalloc((void **) &unfilteredDepth, N * sizeof(float)));

    CUDA_CALL(hipMalloc((void **) &previousFrame.depthMap, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &previousFrame.renderedImage, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &previousFrame.g_vertices, N * sizeof(Vector3f)));
    CUDA_CALL(hipMalloc((void **) &previousFrame.g_normals, N * sizeof(Vector3f)));
    CUDA_CALL(hipMalloc((void **) &previousFrame.globalCameraPose, sizeof(Matrix4f)));

    CUDA_CALL(hipMalloc((void **) &currentFrame.depthMap, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &currentFrame.renderedImage, N * sizeof(float)));
    CUDA_CALL(hipMalloc((void **) &currentFrame.g_vertices, N * sizeof(Vector3f)));
    CUDA_CALL(hipMalloc((void **) &currentFrame.g_normals, N * sizeof(Vector3f)));
    CUDA_CALL(hipMalloc((void **) &currentFrame.globalCameraPose, sizeof(Matrix4f)));

    CUDA_CALL(hipMemcpy(previousFrame.globalCameraPose, currentCameraToWorld.data(), sizeof(Matrix4f), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(currentFrame.globalCameraPose, currentCameraToWorld.data(), sizeof(Matrix4f), hipMemcpyHostToDevice));

    Matrix4f *cuda4fIdentity;
    CUDA_CALL(hipMalloc((void **) &cuda4fIdentity, sizeof(Matrix4f)));
    CUDA_CALL(hipMemcpy(cuda4fIdentity, currentCameraToWorld.data(), sizeof(Matrix4f), hipMemcpyHostToDevice));

    Matrix4f *tmp4fMat_cpu;
    tmp4fMat_cpu = (Matrix4f*) malloc(sizeof(Matrix4f));


    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    Vector3f *g_vertices_host;
    g_vertices_host = (Vector3f *) malloc(N * sizeof(Vector3f));
  
    cv::Mat renderedDepthImg = cv::Mat::zeros(depthFrameHeight, depthFrameWidth, CV_32FC1);


	int i = 0;
	const int iMax = 5;
	while (sensor.processNextFrame() && i < iMax) {
	    // Get current depth frame
		float* depthMap = sensor.getDepth();

		// Copy depth map to current frame, device memory
        CUDA_CALL(hipMemcpy(unfilteredDepth, depthMap, N * sizeof(float), hipMemcpyHostToDevice));

        hipEventRecord(start);
        BilateralFilter::filterDepthmap(unfilteredDepth,currentFrame.depthMap,depthFrameWidth,100,3,depthFrameHeight,N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start,stop);
        printf("Filter depthmap elapsed time : %f ms\n" ,elapsedTime);


        // #### Step 1: Surface measurement
        // It expects the pointers for device memory

        hipEventRecord(start);
        surfaceMeasurement.measureSurface(depthFrameWidth, depthFrameHeight,
                                            currentFrame.g_vertices, currentFrame.g_normals, currentFrame.depthMap,
                                          0);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start,stop);
        printf("Surface measurement elapsed time : %f ms\n" ,elapsedTime);

        ///// Debugging code start
        //// We write out the mesh to file for debugging.
//        Vector3f *g_vertices_host;
//        g_vertices_host = (Vector3f *) malloc(N * sizeof(Vector3f));
//        std::cout << "step 6" << std::endl;
//        CUDA_CALL(hipMemcpy(g_vertices_host, currentFrame.g_vertices, N * sizeof(Vector3f), hipMemcpyDeviceToHost));

//        SimpleMesh currentSM{ currentFrame.g_vertices, depthFrameWidth,depthFrameHeight, sensor.getColorRGBX(), 0.1f };
//        std::stringstream ss1;
//        ss1 << filenameBaseOut << "SM_" << sensor.getCurrentFrameCnt() << ".off";
//        if (!currentSM.writeMesh(ss1.str())) {
//            std::cout << "Failed to write mesh!\nCheck file path!" << std::endl;
//            return -1;
//        }
//        free(g_vertices_host);
        ///// Debugging code  end

		// #### Step 2: Pose Estimation (Using Linearized ICP)
		Matrix4f currentFrameToPreviousFrame = Matrix4f::Identity();
		// Don't do ICP on 1st  frame
		if (i > 0) {
            if (USE_GPU_ICP)  {
                // The arguments should be on device memory
                // The returned pose matrix will be on host memory
                hipEventRecord(start);
                currentFrameToPreviousFrame = optimizer->estimatePose(*cudaDepthIntrinsics, currentFrame, previousFrame, *previousFrame.globalCameraPose);

                hipEventRecord(stop);
                hipEventSynchronize(stop);

                hipEventElapsedTime(&elapsedTime, start,stop);
                printf("ICP elapsed time : %f ms\n" ,elapsedTime);
            }
            else {
                // currentCameraToWorld = optimizer->estimatePose(depthIntrinsics, currentFrame, previousFrame, Matrix4f::Identity());
            }
		}
        //std::cout << "currentFrameToPreviousFrame pose: " << std::endl << currentFrameToPreviousFrame << std::endl;
        currentCameraToWorld = currentFrameToPreviousFrame * currentCameraToWorld;

		//// Step 3:  Volumetric Grid Fusion

		// @TODO: copy  currentCameraToWorld  to gpu
		hipEventRecord(start);

		volumetricGrid.integrateFrame(&currentCameraToWorld,  currentFrame);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start,stop);
        printf("Volumetric fusion elapsed time : %f ms\n" ,elapsedTime);


        // Step 4: Ray-Casting
        surfacePrediction.predict(volumetricGrid,
		        currentFrame.g_vertices,
		        currentFrame.g_normals,
		        currentFrame.renderedImage,
		        currentCameraToWorld,
                depthFrameWidth,
                depthFrameHeight);


		// Step 5: Update trajectory poses
		// Invert the transformation matrix to get the current camera pose.  [Host memory]
        Matrix4f currentCameraPose = currentCameraToWorld.inverse();
		std::cout << "Current camera pose: " << std::endl << currentCameraPose << std::endl;
		estimatedPoses.push_back(currentCameraPose);

		// Update globalCameraPose on device memory
        CUDA_CALL(hipMemcpy(currentFrame.globalCameraPose, currentCameraPose.data(), sizeof(Matrix4f), hipMemcpyHostToDevice));

        // Render the raycast result
        CUDA_CALL(hipMemcpy(renderedDepthImg.data, currentFrame.renderedImage, N * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(g_vertices_host, currentFrame.g_vertices, N * sizeof(Vector3f), hipMemcpyDeviceToHost));
        std::cout << "Saving rendered mesh ..." << std::endl;
        SimpleMesh currentSM{ g_vertices_host, depthFrameWidth,depthFrameHeight, sensor.getColorRGBX(), false,0.1f };
        std::stringstream ss1;
        ss1 << filenameBaseOut << "SM_" << sensor.getCurrentFrameCnt() << ".off";
        if (!currentSM.writeMesh(ss1.str())) {
            std::cout << "Failed to write mesh!\nCheck file path!" << std::endl;
            return -1;
        }

        // Step 7: Update data (e.g. Poses, depth frame etc.) for next frame
		// Update previous frame data
        FrameData tmpFrame = previousFrame;
        previousFrame = currentFrame;
        currentFrame = tmpFrame;

        if(SHOW_MESH) {
            cv::Mat img = cv::Mat::zeros(480, 640, CV_32F);

            std::vector<Vector3f> normals = std::vector<Vector3f>(640 * 480);

            CUDA_CALL(hipMemcpyAsync(normals.data(), previousFrame.g_normals, sizeof(Vector3f) * 640 * 480,
                                      hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());

            std::cout << "Generating img" << std::endl;
            for (int normal_idx = 0; normal_idx < normals.size(); normal_idx++) {
                img.at<float>(normal_idx) = normals[normal_idx].dot((Vector3f(1, 1, 1).normalized()));
            }
            std::cout << "Done." << std::endl;

            cv::namedWindow("Current mesh");
            cv::imshow("Current mesh", img);
            cv::waitKey(0);
            cv::destroyAllWindows();
        }

		// if (i % 5 == 0) {

		if (i > 0) {
		    std::cout << "Saving mesh ..." << std::endl;
            // We write out the mesh to file for debugging.
            std::vector<Vector3f> cpu_vertices = std::vector<Vector3f>(640 * 480);
            CUDA_CALL(hipMemcpyAsync(cpu_vertices.data(),previousFrame.g_vertices,sizeof(Vector3f) * 640 * 480,hipMemcpyDeviceToHost));
            CUDA_CALL(hipDeviceSynchronize());
            SimpleMesh filteredDepthMesh{cpu_vertices.data(),640,480,sensor.getColorRGBX(), 0.1f};
            //SimpleMesh currentDepthMesh{ sensor, currentCameraPose, 0.1f };
            SimpleMesh currentCameraMesh = SimpleMesh::camera(currentCameraPose, 0.0015f);
            SimpleMesh resultingMesh = SimpleMesh::joinMeshes(filteredDepthMesh, currentCameraMesh, Matrix4f::Identity());

            std::stringstream ss;
            ss << filenameBaseOut << sensor.getCurrentFrameCnt() << ".off";
            if (!resultingMesh.writeMesh(ss.str())) {
                std::cout << "Failed to write mesh!\nCheck file path!" << std::endl;
                return -1;
            }
		}

		i++;
	}

	//  Save Volumetric Grid as pointclouds
    std::stringstream ss2, ss3;
    ss2 << filenameBaseOut << "tsdf.ply";
    ss3 << filenameBaseOut << "tsdf.bin";
    volumetricGrid.copyVGFromDeviceToHost();
	volumetricGrid.SaveVoxelGrid2SurfacePointCloud(ss2.str(),  0.2f, 0.0f);
    volumetricGrid.SaveVoxelGrid(ss3.str());

	// Free all pointers
    CUDA_CALL(hipFree(cudaDepthIntrinsics));

    CUDA_CALL(hipFree(previousFrame.depthMap));
    CUDA_CALL(hipFree(previousFrame.g_vertices));
    CUDA_CALL(hipFree(previousFrame.g_normals));
    CUDA_CALL(hipFree(previousFrame.globalCameraPose));

    CUDA_CALL(hipFree(currentFrame.depthMap));
    CUDA_CALL(hipFree(currentFrame.g_vertices));
    CUDA_CALL(hipFree(currentFrame.g_normals));
    CUDA_CALL(hipFree(currentFrame.globalCameraPose));

    free(tmp4fMat_cpu);
    free(g_vertices_host);

    delete optimizer;

	return 0;
}

int main() {
    int result = reconstructRoom();
	return result;
}
